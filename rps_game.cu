#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>

#define NUM_ROUNDS 1000
#define ROCK 0
#define PAPER 1
#define SCISSORS 2

const char *moveName(int move) {
    switch (move) {
        case ROCK: return "Rock";
        case PAPER: return "Paper";
        case SCISSORS: return "Scissors";
        default: return "Invalid";
    }
}

__device__ int counterMostFrequent(int *opponentMoves, int round) {
    int rock = 0, paper = 0, scissors = 0;
    for (int i = 0; i < round; ++i) {
        if (opponentMoves[i] == ROCK) rock++;
        else if (opponentMoves[i] == PAPER) paper++;
        else if (opponentMoves[i] == SCISSORS) scissors++;
    }

    if (rock >= paper && rock >= scissors) return PAPER;
    else if (paper >= scissors) return SCISSORS;
    else return ROCK;
}

__global__ void playGame(int *gpu0Moves, int *gpu1Moves, int *results, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NUM_ROUNDS) return;

    // GPU 0: random
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    gpu0Moves[idx] = hiprand(&state) % 3;

    __syncthreads();

    // GPU 1: pattern-based
    if (idx == 0)
        gpu1Moves[idx] = hiprand(&state) % 3;
    else
        gpu1Moves[idx] = counterMostFrequent(gpu0Moves, idx);

    // Result: -1 = GPU 0 wins, 0 = tie, 1 = GPU 1 wins
    int p0 = gpu0Moves[idx];
    int p1 = gpu1Moves[idx];
    if (p0 == p1) results[idx] = 0; // Ties
    else if ((p0 == ROCK && p1 == SCISSORS) || (p0 == PAPER && p1 == ROCK) || (p0 == SCISSORS && p1 == PAPER))
        results[idx] = -1; // GPU 0 wins
    else
        results[idx] = 1; // GPU 1 wins
}

int main() {
    // Initailise GPUs
    int *gpu0Moves = new int[NUM_ROUNDS];
    int *gpu1Moves = new int[NUM_ROUNDS];
    int *results = new int[NUM_ROUNDS];

    //Cuda
    int *d_gpu0Moves, *d_gpu1Moves, *d_results;
    hipMalloc(&d_gpu0Moves, NUM_ROUNDS * sizeof(int));
    hipMalloc(&d_gpu1Moves, NUM_ROUNDS * sizeof(int));
    hipMalloc(&d_results, NUM_ROUNDS * sizeof(int));

    //Plating games
    playGame<<<(NUM_ROUNDS + 255) / 256, 256>>>(d_gpu0Moves, d_gpu1Moves, d_results, time(NULL));
    hipDeviceSynchronize();

    //Copy memory
    hipMemcpy(gpu0Moves, d_gpu0Moves, NUM_ROUNDS * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(gpu1Moves, d_gpu1Moves, NUM_ROUNDS * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(results,   d_results,   NUM_ROUNDS * sizeof(int), hipMemcpyDeviceToHost);

    // Calculating GPU1 and GPU2 wins and ties
    int win0 = 0, win1 = 0, tie = 0;

    std::ofstream logfile("logs/game_log.txt"); // logs file entry per game
    logfile << "Round, GPU0, GPU1, Result\n";

    for (int i = 0; i < NUM_ROUNDS; ++i) {
        const char *p0 = moveName(gpu0Moves[i]); // GPU0 move
        const char *p1 = moveName(gpu1Moves[i]); // GPU1 move
        const char *resultStr; // final result

        // assiging Result in string
        if (results[i] == -1) {
            resultStr = "GPU 0 Wins";
            win0++;
        } else if (results[i] == 1) {
            resultStr = "GPU 1 Wins";
            win1++;
        } else {
            resultStr = "Tie";
            tie++;
        }

        logfile << i + 1 << ", " << p0 << ", " << p1 << ", " << resultStr << "\n"; // writing in log file
    }

    // Overall summary of the games played
    logfile << "\nSummary:\n";
    logfile << "GPU 0 Wins: " << win0 << "\n";
    logfile << "GPU 1 Wins: " << win1 << "\n";
    logfile << "Ties: " << tie << "\n";
    logfile.close();

    std::cout << "Game complete. Results written to logs/game_log.txt\n";

    // Free the memory
    hipFree(d_gpu0Moves);
    hipFree(d_gpu1Moves);
    hipFree(d_results);
    delete[] gpu0Moves;
    delete[] gpu1Moves;
    delete[] results;

    return 0;
}
